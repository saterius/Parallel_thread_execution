
#include <hip/hip_runtime.h>
#include<stdio.h>
#define N 32
#define BLOCK_SIZE 32

void add(int *X, int *Y, int *Z) {
	for(int i = 0; i < N; i++)
		for(int j = 0; j < N; j++) {
			Z[i*N+j] = X[i*N+j] + Y[i*N+j];
		}
}

__global__ void add_kernel(int *X, int *Y, int *Z){
	int i = threadIdx.x;
	int j = threadIdx.y;

	Z[i*N+j] = X[i*N+j] + Y[i*N+j];
}

int main()
{
	int n;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);


	printf("Input interger: ");
	scanf("%d", &n);

	//Input matrix
	int X[N*N];
	int Y[N*N];

	for(int i = 0; i < N; i++)
		for(int j = 0; j < N; j++) {
			X[i*N+j] = -1;
			Y[i*N+j] = 1;
		}

	//Output matrix
	int Z[N*N];

	int *d_X, *d_Y, *d_Z;
	hipMalloc((void**) &d_X, (N*N)*sizeof(int));
	hipMalloc((void**) &d_Y, (N*N)*sizeof(int));
	hipMalloc((void**) &d_Z, (N*N)*sizeof(int));

	hipMemcpy(d_X, &X, (N*N)*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_Y, &Y, (N*N)*sizeof(int), hipMemcpyHostToDevice);

	dim3 dimGrid(1,1,1);
	dim3 dimBlock(20,20,1);

	hipEventRecord(start);
	add_kernel<<<dimGrid, dimBlock>>>(d_X, d_Y, d_Z);
	hipEventRecord(stop);

	//add(X, Y, Z);

	hipMemcpy(&Z, d_Z, (N*N)*sizeof(int), hipMemcpyDeviceToHost);

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	hipFree(d_X);
	hipFree(d_Y);
	hipFree(d_Z);

	printf("%f ms\n", milliseconds);

	for(int i = 0; i < N; i++) {
		for(int j = 0; j < N; j++) {
			printf("%d ", Z[i*N+j]);
		}
		printf("\n");
	}

	return -1;
}
